
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <random>
#include <iostream>
#include <bits/stdc++.h>
#include <math.h>
#include <fstream>

// #define N_THREADS 
#define N_BLOCKS 1

/*** GPU functions ***/

// Update acceleration of particles
__device__ void get_acc_kernel(double *p, double *m, double *a, double G, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Accleration (x, y, z) for plaent with id tid
  double x = 0;
  double y = 0;
  double z = 0;

  for(int i=0; i<N; i++){
    if(i != tid){
      // Get difference in position of neighboring particle
      double dx = p[0 + i * 3] - p[0 + tid * 3];
      double dy = p[1 + i * 3] - p[1 + tid * 3];
      double dz = p[2 + i * 3] - p[2 + tid * 3];
      // printf("p1-pj1=dx: %e - %e = %e; p2-pj2=dy: %e - %e = %e; p3-pj3=dz: %e - %e = %e\n", p[0 + i * 3], p[0 + tid * 3], dx, p[1 + i * 3], p[1 + tid * 3], dy, p[2 + i * 3],p[2 + tid * 3],dz);

      // Calculate inverse with softening length (0.1) -- Part to account for particles close to eachother
      double inv = pow(pow(dx, 2) + pow(dy, 2) + pow(dz, 2) + pow(0.1, 2), -1.5);
      // printf("inv: %e\n", inv);

      x = x + (m[i] * dx / inv);
      y = y + (m[i] * dy / inv);
      z = z + (m[i] * dz / inv);
      // printf("x: %e; y: %e; z: %e\n", x, y, z);
    }
  }      

  // Adjust with Newton's Gravitational constant
  x = x * G;
  y = y * G;
  z = z * G;
  // printf("x: %e; y: %e; z: %e\n", x, y, z);

  // Assign new x,y,z accelerations to "a"
  a[0 + tid * 3] = x;
  a[1 + tid * 3] = y;
  a[2 + tid * 3] = z;
}

// Update velocity of singular particle (used each half kick)
__device__ void get_vel_kernel(double *v, double *a, double td) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  v[0 + tid * 3] = v[0 + tid * 3] + (a[0 + tid * 3] * td / 2.0);
  v[1 + tid * 3] = v[1 + tid * 3] + (a[1 + tid * 3] * td / 2.0);
  v[2 + tid * 3] = v[2 + tid * 3] + (a[2 + tid * 3] * td / 2.0);

  // printf("v = v + a(td/2): %e = %e + %e(%e/2)\n",v[0 + tid * 3] , v[0 + tid * 3] , a[0 + tid * 3] , td );
  // printf("v = v + a(td/2): %e = %e + %e(%e/2)\n",v[1 + tid * 3] , v[1 + tid * 3] , a[1 + tid * 3] , td );
  // printf("v = v + a(td/2): %e = %e + %e(%e/2)\n",vnum[2 + tid * 3] , v[2 + tid * 3] , a[2 + tid * 3] , td );
}

  // Update position of singular particle (drift)
__device__ void get_pos_kernel(double *p, double *v, double *data, double td, int N, int i) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // new position = position + velocity * td
  p[0 + tid * 3] = p[0 + tid * 3] + (v[0 + tid * 3] * td);
  p[1 + tid * 3] = p[1 + tid * 3] + (v[1 + tid * 3] * td);
  p[2 + tid * 3] = p[2 + tid * 3] + (v[2 + tid * 3] * td);

  // printf("p=p+v*td: %e = %e+%e*%e\n", p[0 + tid * 3], p[0 + tid * 3], v[0 + tid * 3], td);
  // printf("p1=p1+v*td: %e = %e+%e*%e\n", p[1 + tid * 3], p[1 + tid * 3], v[1 + tid * 3], td);
  // printf("p2=p2+v*td: %e = %e+%e*%e\n", p[2 + tid * 3], p[2 + tid * 3], v[2 + tid * 3], td);

  data[N + (i * N * 3 + 3 * tid + 0)] = p[0 + tid * 3];
  data[N + (i * N * 3 + 3 * tid + 1)] = p[1 + tid * 3];
  data[N + (i * N * 3 + 3 * tid + 2)] = p[2 + tid * 3];
  // printf("d0: %e; d1: %e; d2: %e\n", data[N + (i * N * 3 + 3 * tid + 0)], data[N + (i * N * 3 + 3 * tid + 1)], data[N + (i * N * 3 + 3 * tid + 2)]);
  // printf("p0: %e; p1: %e; p2: %e\n", p[0 + tid * 3], p[1 + tid * 3], p[2 + tid * 3]);

  // printf("d0[%d] = p0[%d]; d1[%d] = p1[%d]; d2[%d] = p2[%d]\n", N +(i * N * 3 + 3 * tid + 0), (0 + tid * 3), N + (i * N * 3 + 3 * tid + 1), (1 + tid * 3), N + (i * N * 3 + 3 * tid + 2), (2 + tid * 3));
}

// Run N-body simulation
__global__ void generate_data_kernel(double *p, double * v, double *m, double *a, double *data, int timesteps, double td, int G, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // Get acceleration of particles
  get_acc_kernel(p, m, a, G, N);
  __syncthreads();

  // Loop for number of timesteps --> timestep 0 already complete
  for(int i = 1; i < timesteps; i++){
    // Use leapfrog integration
    // 1) First half kick --> update velocities
    get_vel_kernel(v, a, td);
    __syncthreads();

    // 2) Drift --> update positions
    get_pos_kernel(p, v, data, td, N, i);
    __syncthreads();

    // 3) update acceleration with new positions
    get_acc_kernel(p, m, a, G, N);
    __syncthreads();
    
    // 4) Second half od kick --> update velocities again
    get_vel_kernel(v, a, td);
    __syncthreads();
  }
}

/*** CPU functions ***/

// Returns data (mass and all positions) from N-body simulation
double* n_body(int N, double G, double td, int timesteps) {
  // Array of random starting positions of particles (N x (x,y,z))
  double* particle_pos = new double[N*3];
  double* d_particle_pos;
  // Array of random velocities of particles
  double* particle_vel = new double[N*3];
  double* d_particle_vel;
  // Array of random masses of particles
  double* particle_mass = new double[N];
  double* d_particle_mass;
  // Array of random masses of particles
  double* particle_acc;
  double* d_particle_acc;
  // Array of positions of particles over all timesteps
  double* data = new double[N * 3 * timesteps + N];
  double* d_data;

  // Allocate memory
  particle_acc = (double*)malloc((N*3)* sizeof(double));
  particle_pos = (double*)malloc((N*3)* sizeof(double));
  particle_vel = (double*)malloc((N*3)* sizeof(double));
  particle_mass = (double*)malloc(N * sizeof(double));
  data = (double*)malloc((N * 3 * timesteps + N) * sizeof(double));

  hipMalloc(&d_particle_mass, N * sizeof(double));
  hipMalloc(&d_particle_pos, N * 3 * sizeof(double));
  hipMalloc(&d_particle_vel, N * 3 * sizeof(double));
  hipMalloc(&d_particle_acc, N * 3 * sizeof(double));
  hipMalloc(&d_data, (N * 3 * timesteps + N) * sizeof(double));

  // Fill array of masses of particles
  for(int i=0; i<N; i++){
    particle_mass[i] = rand()/double(RAND_MAX)*1.f+0.f;
    data[i] = particle_mass[i];
  }

  // Fill array of random starting velocities & positions for each particle
  for(int i= 0; i<N; i++){
    if (i % 3 == 1){
      particle_pos[0 + 3 * i] = rand()/double(RAND_MAX)*3.f-2.f;
      particle_pos[1 + 3 * i] = rand()/double(RAND_MAX)*2.f-2.f;
      particle_pos[2 + 3 * i] = rand()/double(RAND_MAX)*2.f-2.f;
    }
    else if (i % 3 == 2){
      particle_pos[0 + 3 * i] = rand()/double(RAND_MAX)*0.f-2.f;
      particle_pos[1 + 3 * i] = rand()/double(RAND_MAX)*2.f-2.f;
      particle_pos[2 + 3 * i] = rand()/double(RAND_MAX)*2.f-2.f;
    }
    else{
      particle_pos[0 + 3 * i] = rand()/double(RAND_MAX)*2.f-2.f;
      particle_pos[1 + 3 * i] = rand()/double(RAND_MAX)*2.f+0.f;
      particle_pos[2 + 3 * i] = rand()/double(RAND_MAX)*2.f-2.f;
    }
    particle_vel[0 + 3 * i] = rand()/double(RAND_MAX)*1.f+0.f;
    particle_vel[1 + 3 * i] = rand()/double(RAND_MAX)*1.f+0.f;
    particle_vel[2 + 3 * i] = rand()/double(RAND_MAX)*1.f+0.f;
    // std::cout <<  " start pos : " << particle_pos[0 + 3*i] << ", " << particle_pos[1 + 3 * i] << ", " << particle_pos[2 + 3 * i] <<std::endl;
    
    // Save initial particle positions
    data[(0 + 3 * i) + N] = particle_pos[0 + 3*i];
    data[(1 + 3 * i) + N] = particle_pos[1 + 3*i];
    data[(2 + 3 * i) + N] = particle_pos[2 + 3*i];
    // std::cout << (0 + 3 * i) + N << " = " << particle_pos[0 + 3*i] <<std::endl;
  }

  // Copy variables from host to device
  hipMemcpy(d_particle_mass, particle_mass, N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_particle_pos, particle_pos, N * 3 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_particle_vel, particle_vel, N * 3 * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_data, data, N * 3 * timesteps * sizeof(double), hipMemcpyHostToDevice);

  // Call GPU kernel to run simulation
  generate_data_kernel<<<N_BLOCKS, N>>>(d_particle_pos, d_particle_vel, d_particle_mass, d_particle_acc, d_data,timesteps, td, G, N);

  // Copy varibles device to host --> maybe just need 
  hipMemcpy(data, d_data, (N * 3 * timesteps +N) * sizeof(double), hipMemcpyDeviceToHost);

  // Free memory
  hipFree(d_particle_pos);
  hipFree(d_particle_vel);
  hipFree(d_particle_acc);
  hipFree(d_particle_mass);
  hipFree(d_data);
  free(particle_pos);
  free(particle_vel);
  free(particle_acc);
  free(particle_mass); 

  // Return particles masses and all positions --> data
  return data;
}


int main(int argc, char** argv) {
  // Number of particles 
  int N = atoi(argv[1]);
  // Newton's Gravitational Constant
  double G = pow(6.67 * 10, -11);
  
  // Start time of simulation
  auto t_start = std::chrono::high_resolution_clock::now();

  //  Set number of timesteps (number of interations for simulation)
  double td = 0.01;
  int timesteps = atoi(argv[2]);

  // Run N-body simulation
  double* data = n_body(  N, G, td, timesteps);

  // End time of simulation
  auto t_end = std::chrono::high_resolution_clock::now();
  // Runtime duration in seconds
  auto total_time = std::chrono::duration_cast<std::chrono::microseconds>(t_end - t_start) * 0.000001;
  std::cout << "Computation Duration: " << total_time.count() << std::endl;

  // Write to output file
  std::ofstream output_file;
  output_file.open("../data/output_cu.txt");
  output_file << "Positions of " << N << " particles over " << timesteps <<" timesteps: \n";
  
  // Write masses
  for(int i=0; i < N; i++){
    if (i == N-1) {
      output_file << data[i] << "\n";
    } else {
      output_file << data[i] << ", ";
    }
  }

  // Write runtime duration
  output_file << total_time.count() << "\n";

  // Write positions
  int curr_step = 0;
  for(int i=N; i < timesteps * N * 3 + N; i++){
    if (curr_step == (N*3)-1) {
      // std::cout << "end i : " << i << std::endl;
      output_file << data[i] << "\n";
      curr_step = 0;
    } else {
      curr_step++;
      // std::cout << "comma i : " << data[i] << std::endl;
      output_file << data[i] << ", ";
    }
  }
  output_file.close();

  // Free data
  free(data);

  return 0;
}
